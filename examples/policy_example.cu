#include "hip/hip_runtime.h"
/*
  ScatterAlloc: Massively Parallel Dynamic Memory Allocation for the GPU.
  http://www.icg.tugraz.at/project/mvp

  Copyright (C) 2012 Institute for Computer Graphics and Vision,
                     Graz University of Technology

  Author(s):  Markus Steinberger - steinberger ( at ) icg.tugraz.at
              Michael Kenzel - kenzel ( at ) icg.tugraz.at

  Permission is hereby granted, free of charge, to any person obtaining a copy
  of this software and associated documentation files (the "Software"), to deal
  in the Software without restriction, including without limitation the rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included in
  all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
  THE SOFTWARE.
*/

#include <hip/hip_runtime.h>

//replace the cuda malloc and free calls
//#define SCATTERALLOC_OVERWRITE_MALLOC 1

//set the template arguments using SCATTERALLOC_HEAPARGS
// pagesize ... byter per page
// accessblocks ... number of superblocks
// regionsize ... number of regions for meta data structur
// wastefactor ... how much memory can be wasted per alloc (multiplicative factor)
// use_coalescing ... combine memory requests of within each warp
// resetfreedpages ... allow pages to be reused with a different size
#define SCATTERALLOC_HEAPARGS 4096, 8, 16, 2, true, false

//include the scatter alloc heap
#include <src/include/scatteralloc/policy_based_heap_impl.cuh>
//#include <src/include/scatteralloc/heap_impl.cuh>

#include <src/include/scatteralloc/utils.h>





#ifdef WIN32
#define WIN32_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#include <conio.h>
#endif

#include <iostream>
#include <stdio.h>

typedef GPUTools::uint32 uint;

void runexample(int cuda_device);


int main(int argc, char** argv)
{
   try
  {
    int cuda_device = argc > 1 ? atoi(argv[1]) : 0;

    hipDeviceProp_t deviceProp;
	  hipGetDeviceProperties(&deviceProp, cuda_device);
    std::cout << "Using device: " << deviceProp.name << std::endl;

	  if( deviceProp.major < 2 ) {
		  std::cerr << "This GPU with Compute Capability " << deviceProp.major 
        << "." << deviceProp.minor <<  " does not meet minimum requirements." << std::endl;
		  std::cerr << "A GPU with Compute Capability >= 2.0 is required." << std::endl;
      return -2;
	  }
  
    std::cout << "start" <<std::endl;
    runexample(cuda_device);
    std::cout << "start" <<std::endl;

    hipDeviceReset();
  }
  catch (const std::exception& e)
  {
    std::cout << e.what()  << std::endl;
    #ifdef WIN32
    while (!_kbhit());
    #endif
    return -1;
  }
  catch (...)
  {
    std::cout << "unknown exception!" << std::endl;
    #ifdef WIN32
    while (!_kbhit());
    #endif
    return -1;
  }

  return 0;
}


__global__ void allocSomething(uint** parray)
{
  parray[threadIdx.x + blockIdx.x*blockDim.x] = new uint[threadIdx.x % 4];
}
__global__ void freeSomething(uint** parray)
{
  delete[] parray[threadIdx.x + blockIdx.x*blockDim.x];
}


void runexample(int cuda_device)
{
  hipSetDevice(cuda_device);

  //init the heap
  initHeap();
  //you can also specify the size of the heap in bytes
  //initHeap(8U*1024U*1024U);

  size_t block = 128;
  size_t grid = 64;

  uint** data;
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc(&data, grid*block*sizeof(uint*)));
  allocSomething<<<grid,block>>>(data);
  SCATTERALLOC_CUDA_CHECKED_CALL(hipDeviceSynchronize());
  freeSomething<<<grid,block>>>(data);
  SCATTERALLOC_CUDA_CHECKED_CALL(hipDeviceSynchronize());
}
