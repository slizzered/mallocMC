#include "hip/hip_runtime.h"
/*
ScatterAlloc: Massively Parallel Dynamic Memory Allocation for the GPU.
http://www.icg.tugraz.at/project/mvp

Copyright (C) 2012 Institute for Computer Graphics and Vision,
Graz University of Technology

Author(s):  Markus Steinberger - steinberger ( at ) icg.tugraz.at
Michael Kenzel - kenzel ( at ) icg.tugraz.at
Carlchristian Eckert - c.eckert ( at ) hzdr.de

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/

#include <hip/hip_runtime.h>

//replace the cuda malloc and free calls
#define SCATTERALLOC_OVERWRITE_MALLOC 1

//set the template arguments using SCATTERALLOC_HEAPARGS
// pagesize ... byter per page
// accessblocks ... number of superblocks
// regionsize ... number of regions for meta data structur
// wastefactor ... how much memory can be wasted per alloc (multiplicative factor)
// use_coalescing ... combine memory requests of within each warp
// resetfreedpages ... allow pages to be reused with a different size
#define SCATTERALLOC_HEAPARGS 4096*1024, 8, 16, 2, true, true
#define ELEMS_PER_SLOT 750

//include the scatter alloc heap
#include <src/include/scatteralloc/heap_impl.cuh>
#include <src/include/scatteralloc/utils.h>

#ifdef WIN32
#define WIN32_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#include <conio.h>
#endif

#include <iostream>
#include <stdio.h>
#include <typeinfo>
#include <vector>

#define CUDA_CHECK(cmd) {hipError_t error = cmd; \
  if(error!=hipSuccess){\
    printf("<%s>:%i ",__FILE__,__LINE__);\
    printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}
/*start kernel, wait for finish and check errors*/
#define CUDA_CHECK_KERNEL_SYNC(...) __VA_ARGS__;CUDA_CHECK(hipDeviceSynchronize())

struct nullstream : std::ostream {
  nullstream() : std::ostream(0) { }
};
 
typedef GPUTools::uint32 uint;
typedef int8_t allocElem_t;

bool run_heap_verification(const int cuda_device, const int verbosity);
void parse_cmdline(const int argc, char**argv,int* verbosity);


/*
 * @brief will do a basic verification of scatterAlloc.
 * @param argv if -q or --quiet is supplied as a
 *        command line argument, verbosity will be reduced
 * 
 * @return will return 0 if the verification was successful,
 *         otherwise returns 1
 */
int main(int argc, char** argv){
  bool correct = false;
  try
  {
    int cuda_device = argc > 1 ? atoi(argv[1]) : 0;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, cuda_device);
    std::cout << "Using device: " << deviceProp.name << std::endl;

    if( deviceProp.major < 2 ) {
      std::cerr << "This GPU with Compute Capability " << deviceProp.major 
        << "." << deviceProp.minor <<  " does not meet minimum requirements." << std::endl;
      std::cerr << "A GPU with Compute Capability >= 2.0 is required." << std::endl;
      return -2;
    }

    int verbosity = 2;
    parse_cmdline(argc,argv,&verbosity);
    correct = run_heap_verification(cuda_device,verbosity);

    hipDeviceReset();
  }
  catch (const std::exception& e)
  {
    std::cout << e.what()  << std::endl;
#ifdef WIN32
    while (!_kbhit());
#endif
    return -1;
  }
  catch (...)
  {
    std::cout << "unknown exception!" << std::endl;
#ifdef WIN32
    while (!_kbhit());
#endif
    return -1;
  }

  if(correct){
    std::cout << "\033[0;32mverification successful ✔\033[0m" << std::endl;
    return 0;
  }else{
    std::cerr << "\033[0;31mverification failed\033[0m" << std::endl;
    return 1;
  }
}


/**
 * @brief will parse command line arguments
 *
 * @param argc argc from main()
 * @param argv argv from main()
 * @param verbosity will be filled with the supplied verbosity
 */
void parse_cmdline(const int argc, char**argv,int* verbosity){
  std::vector<std::pair<std::string, std::string> > parameters;
  //Parse Commandline
  for (int i = 1; i < argc; ++i) {

    char* pos = strtok(argv[i], "=");
    std::pair < std::string, std::string > p(std::string(pos), std::string(""));
    pos = strtok(NULL, "=");
    if (pos != NULL) {
      p.second = std::string(pos);
    }
    parameters.push_back(p);
  }
  for (unsigned i = 0; i < parameters.size(); ++i) {
    std::pair < std::string, std::string > p = parameters.at(i);

    if (p.first == "-q" || p.first == "--quiet") {
      *verbosity = 1;
    }

  }
}

/**
 * @brief checks on a per thread basis, if the values
 *        written during allocation are still the same.
 *        Also calculates the sum over all allocated
 *        values for a more in-depth verification
 * @param data the data to verify
 * @param counter should be initialized with 0 and will
 *        be used to count how many verifications were
 *        already done
 * @param globalSum will be filled with the sum over all
 *        allocated values in the structure
 * @param nSlots the size of the datastructure
 * @param correct should be initialized with 1.
 *        Will change to 0, if there was a value that didn't match
 */
__global__ void check_content(
    allocElem_t** data,
    unsigned long long *counter,
    unsigned long long* globalSum, 
    const size_t nSlots,
    int* correct
    ){

  unsigned long long sum=0;
  while(true){
    size_t pos = atomicAdd(counter,1);
    if(pos >= nSlots){break;}
    const size_t offset = pos*ELEMS_PER_SLOT;
    for(size_t i=0;i<ELEMS_PER_SLOT;++i){
      if (static_cast<allocElem_t>(data[pos][i]) != static_cast<allocElem_t>(offset+i)){
        //printf("\nError in Kernel: data[%llu][%llu] is %#010x (should be %#010x)\n",
        //    pos,i,static_cast<allocElem_t>(data[pos][i]),allocElem_t(offset+i));
        atomicAnd(correct,0);
      }
      sum += static_cast<unsigned long long>(data[pos][i]);
    }
  }
  atomicAdd(globalSum,sum);
}


/**
 * @brief checks on a per thread basis, if the values
 *        written during allocation are still the same.
 * @param data the data to verify
 * @param counter should be initialized with 0 and will
 *        be used to count how many verifications were
 *        already done
 * @param nSlots the size of the datastructure
 * @param correct should be initialized with 1.
 *        Will change to 0, if there was a value that didn't match
 */
__global__ void check_content_fast(
    allocElem_t** data,
    unsigned long long *counter,
    const size_t nSlots,
    int* correct
    ){

  int c = 1;
  while(true){
    size_t pos = atomicAdd(counter,1);
    if(pos >= nSlots){break;}
    const size_t offset = pos*ELEMS_PER_SLOT;
    for(size_t i=0;i<ELEMS_PER_SLOT;++i){
      if (static_cast<allocElem_t>(data[pos][i]) != static_cast<allocElem_t>(offset+i)){
        c=0;
      }
    }
  }
  atomicAnd(correct,c);
}


/**
 * @brief allocate a lot of small arrays, each having
 *        the size ELEMS_PER_SLOT. Each element will
 *        be filled with a number that is related to
 *        its position in the datastructure.
 * @param data the datastructure to allocate
 * @param counter should be initialized with 0 and will
 *        hold, how many allocations were done
 * @param globalSum will hold the sum of all values over all
 *        allocated structures (for verification purposes)
 */
__global__ void allocAll(
    allocElem_t** data, 
    unsigned long long* counter, 
    unsigned long long* globalSum
    ){

  unsigned long long sum=0;
  while(true){
    allocElem_t* p = new allocElem_t[ELEMS_PER_SLOT];
    if(p == NULL) break;

    size_t pos = atomicAdd(counter,1);
    const size_t offset = pos*ELEMS_PER_SLOT;
    for(size_t i=0;i<ELEMS_PER_SLOT;++i){
      p[i] = static_cast<allocElem_t>(offset + i);
      sum += static_cast<unsigned long long>(p[i]);
    }
    data[pos] = p;
  }

  atomicAdd(globalSum,sum);
}


/**
 * @brief free all the values again
 *
 * @param data the datastructure to free
 * @param counter should be an empty space on device memory, 
 *        counts how many elements were freed
 * @param max the maximum number of elements to free
 */
__global__ void deallocAll(
    allocElem_t** data,
    unsigned long long* counter,
    const size_t nSlots
    ){

  while(true){
    size_t pos = atomicAdd(counter,1);
    if(pos >= nSlots) break;
    delete data[pos];
  }
}


/**
 * @brief damages one element in the data, so you 
 *        can see if your checks actually work
 *
 * @param data the datastructure to damage
 */
__global__ void damageElement(allocElem_t** data){
  data[1][0] = static_cast<allocElem_t>(5*ELEMS_PER_SLOT - 1);
}


/**
 * @brief wrapper function to allocate some memory on the device
 *        with scatterAlloc. Returns the number of created elements as well
 *        as the sum of these elements
 *
 * @param d_testData the datastructure which will hold 
 *        pointers to the created elements
 * @param h_nSlots will be filled with the number of elements
 *        that were allocated
 * @param h_sum will be filled with the sum of all elements created
 * @param blocks the size of the CUDA grid
 * @param threads the number of CUDA threads per block
 */
void allocate(
    allocElem_t** d_testData, 
    unsigned long long* h_nSlots, 
    unsigned long long* h_sum,
    const unsigned blocks,
    const unsigned threads
    ){

  std::cout << "allocating on device...";

  unsigned long long zero = 0;
  unsigned long long *d_sum;
  unsigned long long *d_nSlots;

  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc((void**) &d_sum,sizeof(unsigned long long)));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc((void**) &d_nSlots, sizeof(unsigned long long)));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(d_sum,&zero,sizeof(unsigned long long),hipMemcpyHostToDevice));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(d_nSlots,&zero,sizeof(unsigned long long),hipMemcpyHostToDevice));

  CUDA_CHECK_KERNEL_SYNC(allocAll<<<blocks,threads>>>(d_testData,d_nSlots,d_sum));

  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(h_sum,d_sum,sizeof(unsigned long long),hipMemcpyDeviceToHost));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(h_nSlots,d_nSlots,sizeof(unsigned long long),hipMemcpyDeviceToHost));
  hipFree(d_sum);
  hipFree(d_nSlots);
  std::cout << "done" << std::endl;
}


/**
 * @brief wrapper function to verify some allocated memory on the device
 *
 * @param d_testData the datastructure which holds 
 *        pointers to the elements you want to verify
 * @param nSlots the size of d_testData
 * @param blocks the size of the CUDA grid
 * @param threads the number of CUDA threads per block
 * @return true if the verification was successful, false otherwise
 */
bool verify(
    allocElem_t **d_testData,
    const unsigned long long nSlots,
    const unsigned blocks,
    const unsigned threads
    ){

  std::cout << "verifying on device... ";

  const unsigned long long zero = 0;
  int  h_correct = 1;
  int* d_correct;
  unsigned long long *d_sum;
  unsigned long long *d_counter;
  
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc((void**) &d_sum, sizeof(unsigned long long)));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc((void**) &d_counter, sizeof(unsigned long long)));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc((void**) &d_correct, sizeof(int)));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(d_sum,&zero,sizeof(unsigned long long),hipMemcpyHostToDevice));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(d_counter,&zero,sizeof(unsigned long long),hipMemcpyHostToDevice));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(d_correct,&h_correct,sizeof(int),hipMemcpyHostToDevice));

  // can be replaced by a call to check_content_fast, 
  // if the gaussian sum (see below) is not used and you 
  // want to be a bit faster
  CUDA_CHECK_KERNEL_SYNC(check_content<<<blocks,threads>>>(
        d_testData,
        d_counter,
        d_sum,
        static_cast<size_t>(nSlots),
        d_correct
        ));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(&h_correct,d_correct,sizeof(int),hipMemcpyDeviceToHost));

  // This only works, if the type "allocElem_t"
  // can hold all the IDs (usually unsigned long long)
  /*
  std::cout << "verifying on host...";
  unsigned long long h_sum, h_counter;
  unsigned long long gaussian_sum = (ELEMS_PER_SLOT*nSlots * (ELEMS_PER_SLOT*nSlots-1))/2;
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(&h_sum,d_sum,sizeof(unsigned long long),hipMemcpyDeviceToHost));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(&h_counter,d_counter,sizeof(unsigned long long),hipMemcpyDeviceToHost));
  if(gaussian_sum != h_sum){
    std::cerr << "\nGaussian Sum doesn't match: is " << h_sum;
    std::cerr << " (should be " << gaussian_sum << ")" << std::endl;
    h_correct=false;
  }
  if(nSlots != h_counter-(blocks*threads)){
    std::cerr << "\nallocated number of elements doesn't match: is " << h_counter;
    std::cerr << " (should be " << nSlots << ")" << std::endl;
    h_correct=false;
  }
  */

  if(h_correct){
    std::cout << "done" << std::endl;
  }else{
    std::cerr << "failed" << std::endl;
  }

  hipFree(d_correct);
  hipFree(d_sum);
  hipFree(d_counter);
  return static_cast<bool>(h_correct);
}


/**
 * @brief verify that the heap actually holds the 
 *        correct values without corrupting them
 * @param cuda_device the index of 
 *        the graphics card to use
 * @return true if the verification was successful,
 *         false otherwise
 */
bool run_heap_verification(const int cuda_device, const int verbosity){
  hipSetDevice(cuda_device);
  hipSetDeviceFlags(hipDeviceMapHost);

  const unsigned blocks         = 64; 
  const unsigned threads        = 128;
  const size_t heapSize         = size_t(1024U*1024U*1024U) * size_t(4U); //4GB
  const size_t slotSize         = sizeof(allocElem_t)*ELEMS_PER_SLOT;
  const size_t nPointers        = ceil(static_cast<float>(heapSize) / slotSize);
  const size_t maxSlots         = heapSize/slotSize;
  const size_t maxSpace         = maxSlots*slotSize + nPointers*sizeof(allocElem_t*);
  bool correct                  = true;
  const unsigned long long zero = 0;


  std::cout << "ScatterAlloc:       " << "page     sblock region waste coalesc reset" << std::endl;
  printf(      "                    %d  %d      %d     %d     %d       %d\n",SCATTERALLOC_HEAPARGS);
  std::cout << "Gridsize:              " << blocks << std::endl;
  std::cout << "Blocksize:             " << threads << std::endl;
  std::cout << "Allocated elements:    " << ELEMS_PER_SLOT << " x " << sizeof(allocElem_t);
  std::cout << "   Byte (" << slotSize << " Byte)" << std::endl;
  std::cout << "Heap:                  " << heapSize << " Byte";
  std::cout << " (" << heapSize/pow(1024,2) << " MByte)" << std::endl; 
  std::cout << "max space w/ pointers: " << maxSpace << " Byte";
  std::cout << " (" << maxSpace/pow(1024,2) << " MByte)" << std::endl;
  std::cout << "maximum of elements:   " << maxSlots << std::endl;

  // initializing the heap
  initHeap(heapSize); 
  allocElem_t** d_testData;
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc((void**) &d_testData, nPointers*sizeof(allocElem_t*)));

  // allocating with scatterAlloc
  unsigned long long nAllocSlots = 0;
  unsigned long long sumAllocElems = 0;
  allocate(d_testData,&nAllocSlots,&sumAllocElems,blocks,threads);

  std::cout << "allocated elements:    " << nAllocSlots;
  const float allocFrac = static_cast<float>(nAllocSlots)*100/maxSlots;
  std::cout << " (" << allocFrac << "%)" << std::endl;
  const size_t wasted = heapSize - static_cast<size_t>(nAllocSlots) * slotSize;
  std::cout << "wasted heap space:     " << wasted << " Byte";
  std::cout << " (" << wasted/pow(1024,2) << " MByte)" << std::endl;

  // verifying on device
  correct = correct && verify(d_testData,nAllocSlots,blocks,threads);

  // damaging one cell
  std::cout << "damaging of element... ";
  CUDA_CHECK_KERNEL_SYNC(damageElement<<<1,1>>>(d_testData));
  std::cout << "done" << std::endl;

  // verifying on device 
  // THIS SHOULD FAIL (damage was done before!). Therefore, we must inverse the logic
  correct = correct && !verify(d_testData,nAllocSlots,blocks,threads);

  // release all memory
  std::cout << "deallocation...        ";
  unsigned long long* d_dealloc_counter;
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMalloc((void**) &d_dealloc_counter, sizeof(unsigned long long)));
  SCATTERALLOC_CUDA_CHECKED_CALL(hipMemcpy(d_dealloc_counter,&zero,sizeof(unsigned long long),hipMemcpyHostToDevice));
  CUDA_CHECK_KERNEL_SYNC(deallocAll<<<blocks,threads>>>(d_testData,d_dealloc_counter,static_cast<size_t>(nAllocSlots)));
  hipFree(d_dealloc_counter);
  hipFree(d_testData);

  std::cout << "done "<< std::endl;
  return correct;
}
